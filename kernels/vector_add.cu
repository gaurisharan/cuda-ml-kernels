#include <iostream>
#include <hip/hip_runtime.h>

__global__ void vectorAddKernel(float* A, float* B, float* C, int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx < N) {
        C[idx] = A[idx] + B[idx];
    }
}

int main() {
    const int N = 10;
    float A[N], B[N], C[N];
    for(int i=0;i<N;i++) { A[i]=i; B[i]=2*i; }

    float *d_A, *d_B, *d_C;
    size_t size = N * sizeof(float);

    hipMalloc(&d_A, size); hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
    hipMalloc(&d_B, size); hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);
    hipMalloc(&d_C, size);

    int threadsPerBlock = 256;
    int blocks = (N + threadsPerBlock - 1) / threadsPerBlock;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    vectorAddKernel<<<blocks, threadsPerBlock>>>(d_A, d_B, d_C, N);

    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);

    for(int i=0;i<N;i++)
        std::cout << C[i] << " ";
    std::cout << std::endl;

    std::cout << "vector_add kernel execution time: " << milliseconds << " ms" << std::endl;

    hipFree(d_A); hipFree(d_B); hipFree(d_C);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    return 0;
}
// This code performs vector addition on the GPU using CUDA.
// It initializes two vectors A and B, allocates memory on the GPU, copies the vectors to the GPU, launches a kernel to perform the addition, and then copies the result back to the host.
// Finally, it prints the result and frees the allocated GPU memory.

//Output obtained-
// 0 3 6 9 12 15 18 21 24 27 
// vector_add kernel execution time: 1.01581 ms