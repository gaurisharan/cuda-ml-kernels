#include <iostream>
#include <hip/hip_runtime.h>

__global__ void matMulKernel(float* A, float* B, float* C, int N) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if(row < N && col < N) {
        float sum = 0;
        for(int i = 0; i < N; ++i)
            sum += A[row * N + i] * B[i * N + col];
        C[row * N + col] = sum;
    }
}

void matMul(float* A, float* B, float* C, int N) {
    float *d_A, *d_B, *d_C;
    size_t size = N * N * sizeof(float);

    hipMalloc(&d_A, size); hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
    hipMalloc(&d_B, size); hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);
    hipMalloc(&d_C, size);

    dim3 threadsPerBlock(16, 16);
    dim3 blocks((N + 15)/16, (N + 15)/16);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    matMulKernel<<<blocks, threadsPerBlock>>>(d_A, d_B, d_C, N);

    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);
    hipFree(d_A); hipFree(d_B); hipFree(d_C);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    std::cout << "intro kernel execution time: " << milliseconds << " ms" << std::endl;
}

int main() {
    const int N = 4;
    float A[N*N], B[N*N], C[N*N];

    for (int i = 0; i < N*N; i++) {
        A[i] = i + 1;
        B[i] = (i + 1) * 2;
    }

    matMul(A, B, C, N);

    std::cout << "Result matrix:\n";
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++)
            std::cout << C[i*N + j] << " ";
        std::cout << "\n";
    }

    return 0;
}
// This code performs matrix multiplication on the GPU using CUDA.
// It initializes two matrices A and B, performs the multiplication in a kernel, and then copies the result back to the host.
// Finally, it prints the result matrix to the console.

// Output obtained-
// intro kernel execution time: 1.26976 ms
// Result matrix:
// 180 200 220 240 
// 404 456 508 560
// 628 712 796 880
// 852 968 1084 1200
